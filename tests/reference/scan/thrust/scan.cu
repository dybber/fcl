#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <algorithm>
#include <cstdlib>

int main(void)
{
  unsigned int size = 4096*4096;
  thrust::host_vector<int32_t> input_host(size);
  for(int i=0; i < size; i++){
    input_host[i] = i;
  }
  //std::generate(input_host.begin(), input_host.end(), rand);
  for(int i=0; i < 100; i++){
    printf("%d,", input_host[i]);
  }
  printf("\n");

  
  // transfer to device and compute sum
  thrust::device_vector<int32_t> input_device = input_host;
  thrust::device_vector<int32_t> output_device(size);

  thrust::plus<int32_t> binary_op;
  thrust::exclusive_scan(input_device.begin(), input_device.end(), output_device.begin(), 0, binary_op); // in-place scan

  thrust::host_vector<int32_t> output_host = output_device;
  for(int i=0; i < 100; i++){
    printf("%d,", output_host[i]);
  }
  printf("\n");
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  for(int i=0; i < 100; i++){
    //int x = thrust::reduce(input_device.begin(), input_device.end(), 0, binary_op);
    thrust::exclusive_scan(input_device.begin(), input_device.end(), output_device.begin(), 0, binary_op); // in-place scan
  }
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsedTime; 
  hipEventElapsedTime(&elapsedTime , start, stop);
  printf("Avg. time is %f ms\n", elapsedTime/100);
  
  return 0;
}
